#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/mil_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void MaxMILForward(const int nthreads,
    const Dtype* const bottom_data, const Dtype* const bottom_label, 
        const int instance_, Dtype* top_data, Dtype* top_label, int* mask) {
    int mindex = 0;
    CUDA_KERNEL_LOOP(index, nthreads) {
        for (int i = 0;i<instance_;i++){
            //LOG(INFO) << "bottom_data[" << i << "*" << class_ << "+" << c << "]=" <<  i*class_ + c;
            mindex = i*nthreads + index;
            if(bottom_data[mindex] > top_data[index]){
                top_data[index] = bottom_data[mindex]; // set data
                top_label[index] = bottom_label[index];
       /*             if (bottom_label[i] != top_label[i]){ // check if label consistency in batch
                        LOG(FATAL) << "Label in a batch cannot be different";
                    }*/
                mask[index] = mindex;
            }
        }
    }
}


template <typename Dtype>
void MILLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* bottom_label = bottom[1]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  Dtype* top_label = top[1]->mutable_gpu_data();
  
  const int top_count = top[0]->count();
  caffe_gpu_set(top_count, Dtype(-FLT_MAX), top_data);

  int* mask = max_idx_.mutable_gpu_data(); 
  // We'll output the mask to top[1] if it's of size >1.
  switch (this->layer_param_.mil_param().pool()) {
  case MILParameter_PoolMethod_MAX:
    // NOLINT_NEXT_LINE(whitespace/operators)
    MaxMILForward<Dtype><<<CAFFE_GET_BLOCKS(class_), CAFFE_CUDA_NUM_THREADS>>>(
        class_, bottom_data, bottom_label, instance_, top_data, top_label, mask);
    break;
  case MILParameter_PoolMethod_AVE:
      NOT_IMPLEMENTED;
    break;
  case MILParameter_PoolMethod_STOCHASTIC:
      NOT_IMPLEMENTED;
    break;
  default:
    LOG(FATAL) << "Unknown pooling method.";
  }
  CUDA_POST_KERNEL_CHECK;
}


template <typename Dtype>
__global__ void MaxPoolBackward(const int nthreads, const int instance,
        const int* const mask, const Dtype* const top_diff,
        Dtype* const bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
      bottom_diff[mask[index]] = top_diff[index];
  }
}

template <typename Dtype>
void MILLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int count = bottom[0]->count();
  caffe_gpu_set(count, Dtype(0.), bottom_diff);
  const int* mask = max_idx_.gpu_data();
  switch (this->layer_param_.mil_param().pool()) {
  case MILParameter_PoolMethod_MAX:
    // NOLINT_NEXT_LINE(whitespace/operators)
    MaxPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(class_), CAFFE_CUDA_NUM_THREADS>>>(
        class_, instance_, mask, top_diff, bottom_diff);
    break;
  case MILParameter_PoolMethod_AVE:
      NOT_IMPLEMENTED;
    break;
  case MILParameter_PoolMethod_STOCHASTIC:
      NOT_IMPLEMENTED;
    break;
  default:
    LOG(FATAL) << "Unknown pooling method.";
  }
  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_LAYER_GPU_FUNCS(MILLayer);


}  // namespace caffe
